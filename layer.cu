#include "hip/hip_runtime.h"
#include "layer.h"
using namespace std;

Layer::Layer(int filter_size, int featuremaps, int output_size) {
    this->filter_size = filter_size;
    this->featuremaps = featuremaps;
    this->output_size = output_size;
    this->output = NULL;
    this->preact = NULL;
    this->bias = NULL;
    this->weight = NULL;
    this->error = NULL;

    vector<float> h_bias(featuremaps);
    vector<vector<float>> h_weight(featuremaps, vector<float>(filter_size));

    // define the bias and the weights as random values between -0.5 and 0.5
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> dis(-0.5f, 0.5f);

    for (int i=0; i<featuremaps; i++) {
        h_bias[i] = dis(gen);
        for (int j=0; j<filter_size; j++) {
            h_weight[i][j] = dis(gen);
        }
    }

    // make space in GPU memory
    hipMalloc(&output, sizeof(float) * output_size);
    hipMalloc(&preact, sizeof(float) * output_size);
    hipMalloc(&bias, sizeof(float) * featuremaps);
    hipMalloc(&weight, sizeof(float) * filter_size * featuremaps);
    hipMalloc(&der_output, sizeof(float) * output_size);
    hipMalloc(&der_preact, sizeof(float) * output_size);
    hipMalloc(&der_weight, sizeof(float) * filter_size * featuremaps);
    hipMalloc(&error, sizeof(float));

    // copy weights to GPU memory
    hipMemcpy(bias, h_bias.data(), sizeof(float) * featuremaps, hipMemcpyHostToDevice);
    hipMemcpy(weight, h_weight.data(), sizeof(float) * filter_size * featuremaps, hipMemcpyHostToDevice);
    hipMemset(error, 0, sizeof(float));
}
Layer::~Layer()
{
    hipFree(output);
    hipFree(preact);
    hipFree(bias);
    hipFree(weight);
    hipFree(der_output);
    hipFree(der_preact);
    hipFree(der_weight);
    hipFree(error);
}

// Send data one row from dataset to the GPU
void Layer::setOutput(float *data)
{
    hipMemcpy(output, data, sizeof(float) * output_size, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
    hipMemset(output, 0x00, sizeof(float) * output_size);
    hipMemset(preact, 0x00, sizeof(float) * output_size);
}

// clear results of back propagation
void Layer::bp_clear()
{
    hipMemset(der_output, 0x00, sizeof(float) * output_size);
    hipMemset(der_preact, 0x00, sizeof(float) * output_size);
    hipMemset(der_weight, 0x00, sizeof(float) * filter_size * featuremaps);
    hipMemset(error, 0, sizeof(float));
}



// this is a sigmoid activation function
__device__ float step_function(float v)
{
    return 1 / (1 + exp(-v));
}

__global__ void apply_step_function(float *input, float *output, const int N)
{
//    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
//    const int size = blockDim.x * gridDim.x;
//
//    for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
//        output[idx] = step_function(input[idx]);
//    }
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = blockIdx.x;
    int index = (idx_x * blockDim.y * gridDim.x) + (idx_y * gridDim.x) + idx_z;
    output[index] = step_function(input[index]);

}

__global__ void euclidianNorm(int n, float *x, float *result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        atomicAdd(result, x[idx] * x[idx]);
    }
    __syncthreads();
    if (idx == 0) {
        *result = sqrt(*result);
    }
}

__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
//    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
//    const int size = blockDim.x * gridDim.x;
//
//    for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
//        err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
//    }

    int idx_x = threadIdx.x;
    err[idx_x] = ((Y == idx_x ? 1.0f : 0.0f) - output[idx_x]);
}

__global__ void apply_grad(float *output, float *grad, const int N)
{
    // Adds a fraction of the derived weight to the original weight

    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;
    int indx_block = blockIdx.x;
    int index = (idx_z * blockDim.y * blockDim.x * gridDim.x) + (idx_y * blockDim.x * gridDim.x) + (idx_x * gridDim.x) + indx_block;
    output[index] += 1.0E-01f * grad[index];
}

__global__ void fp_preact_c1(float input[28][28], float preact[6][24][24], float weight[6][5][5])
{
    // For 1 Block of 24x24 threads, every thread calculates 6 output values
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;

    if (idx_x < 24 && idx_y < 24) {
        for (int idx_z=0; idx_z < 6; idx_z++) {
            float sum = 0.0f;
            for (int i = 0; i < 5; ++i) {
                for (int j = 0; j < 5; ++j) {
                    sum += input[idx_x + i][idx_y + j] * weight[idx_z][i][j];
                }
            }
            preact[idx_z][idx_x][idx_y] = sum;
        }
    }
}

__global__ void fp_bias_c1(float preact[6][24][24], float bias[6])
{
    // 6 Blocks of 24x24 threads, there is one thread for every output value
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = blockIdx.x;
    preact[idx_z][idx_x][idx_y] += bias[idx_z];
}

__global__ void fp_preact_s1(float input[6][24][24], float preact[6][6][6], float weight[1][4][4])
{
    // For 1 Block of 24x24 threads, every thread calculates multiple output values
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;

    if (idx_x < 6 && idx_y < 6) {
        for (int idx_z = 0; idx_z < 6; idx_z++) {
            float sum = 0.0f;
            for (int i = 0; i < 4; ++i) {
                for (int j = 0; j < 4; ++j) {
                    sum += weight[0][i][j] * input[idx_z][idx_x * 4 + i][idx_y * 4 + j];
                }
            }
            preact[idx_z][idx_x][idx_y] = sum;
        }
    }
}

__global__ void fp_bias_s1(float preact[6][6][6], float bias[1])
{
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = blockIdx.x;
    preact[idx_z][idx_x][idx_y] += bias[idx_z];
}

__global__ void fp_preact_f(float input[6][6][6], float preact[10], float weight[10][6][6][6])  // todo: denk eens na dommerikken!!!
{
    // For 1 Block of 6x6x6 threads, every thread calculates multiple output values
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;

    if (idx_x < 6 && idx_y < 6 && idx_z < 6) {
        for (int i = 0; i < 10; i++) {
            float sum = weight[i][idx_x][idx_y][idx_z] * input[idx_x][idx_y][idx_z];
            atomicAdd(&preact[i], sum);
        }
    }

}

__global__ void fp_bias_f(float preact[10], float bias[10])
{
    int idx_x = threadIdx.x;
    preact[idx_x] += bias[idx_x];
}

__global__ void bp_weight_f(float d_weight[10][6][6][6], float d_preact[10], float p_output[6][6][6])
{
    // 10 Block of 6x6x6 threads, every thread calculates one weight value
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;
    int idx_block = blockIdx.x;

    if (idx_x < 6 && idx_y < 6 && idx_z < 6) {
        d_weight[idx_block][idx_x][idx_y][idx_z] = d_preact[idx_block] * p_output[idx_x][idx_y][idx_z];

    }
}

__global__ void bp_bias_f(float bias[10], float d_preact[10])
{

    // 1 block of 10 threads, every thread calculates one bias by taking 10% of the predicted output
    int idx = threadIdx.x;
    bias[idx] += 1.0E-01f * d_preact[idx];
}

__global__ void bp_output_s1(float d_output[6][6][6], float n_weight[10][6][6][6], float nd_preact[10])
{
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;
    int idx_block = blockIdx.x;

    atomicAdd(&d_output[idx_x][idx_y][idx_z], n_weight[idx_block][idx_x][idx_y][idx_z] * nd_preact[idx_block]);
}

__global__ void bp_preact_s1(float d_preact[6][6][6], float d_output[6][6][6], float preact[6][6][6])
{
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;
    const float output = step_function(preact[idx_x][idx_y][idx_z]);
    // we nemen de afgeleide van de derivate output
    d_preact[idx_x][idx_y][idx_z] = d_output[idx_x][idx_y][idx_z] * output * (1 - output);
}

__global__ void bp_weight_s1(float d_weight[1][4][4], float d_preact[6][6][6], float p_output[6][24][24])
{
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;

    if (idx_x < 4 && idx_y < 4) {
        float sum = 0.0f;
        for (int i4 = 0; i4 < 6; ++i4) {
            for (int i5 = 0; i5 < 6; ++i5) {
                for (int i6 = 0; i6 < 6; ++i6) {
                    sum += d_preact[i4][i5][i6] * p_output[i4][i5 * 4 + idx_x][i6 * 4 + idx_y];
                }
            }
        }
        atomicAdd(&d_weight[0][idx_x][idx_y], sum);
    }
}

__global__ void bp_bias_s1(float bias[1], float d_preact[6][6][6])
{
    // 1 Block of 6x6x6 threads, every thread does one calculation
    const float d = pow(6.0f, 3.0f);

    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;

    atomicAdd(&bias[0], 1.0E-01f * d_preact[idx_x][idx_y][idx_z] / d);
}

__global__ void bp_output_c1(float d_output[6][24][24], float n_weight[1][4][4], float nd_preact[6][6][6])
{
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;
    int idx_z = threadIdx.z;

    for (int i1=0; i1 <4; i1++) {
        for (int i2=0; i2<4; i2++) {
            d_output[idx_x][idx_y*4 +i1][idx_z*4 + i2] += n_weight[0][i1][i2] * nd_preact[idx_x][idx_y][idx_z];
        }
    }
}

__global__ void bp_preact_c1(float d_preact[6][24][24], float d_output[6][24][24], float preact[6][24][24])
{
    int idx_x = blockIdx.x;
    int idx_y = threadIdx.x;
    int idx_z = threadIdx.y;
    const float output = step_function(preact[idx_x][idx_y][idx_z]);
    // we nemen de afgeleide van de derivate output
    d_preact[idx_x][idx_y][idx_z] = d_output[idx_x][idx_y][idx_z] * output * (1 - output);
}

__global__ void bp_weight_c1(float d_weight[6][5][5], float d_preact[6][24][24], float p_output[28][28])
{
    int idx_x = blockIdx.x;
    int idx_y = threadIdx.x;
    int idx_z = threadIdx.y;
    const float d = pow(24.0f, 2.0f);
    float sum = 0.0f;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            sum += d_preact[idx_x][idx_y + i][idx_z + j] * p_output[idx_y + i][idx_z + j];
        }
    }
    d_weight[idx_x][idx_y][idx_z] = sum;

}

__global__ void bp_bias_c1(float bias[6], float d_preact[6][24][24])
{
    const float d = pow(6.0f, 3.0f);

    int idx_x = blockIdx.x;
    int idx_y = threadIdx.x;
    int idx_z = threadIdx.y;

    atomicAdd(&bias[idx_x], 1.0E-01f * d_preact[idx_x][idx_y][idx_z] / d);
}
